#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
#include "virus.h"
#include "community.h"
#include "../random.cuh"

#include <stdio.h>

__device__ void reduce(double *bl)
{
	int i = blockDim.x * blockDim.y / 2;
	while (i != 0)
	{
		int tid = threadIdx.x + threadIdx.y * blockDim.x;
		if (tid < i)
			bl[tid] += bl[tid + i];
		__syncthreads();
		i /= 2;
	}
	__syncthreads();
}

void runDay(SimulationData *sd, int day)
{
	SimulationData *gsd;
	hipMalloc((void **)&gsd, sizeof(SimulationData));
	hipMemcpy(gsd, sd, sizeof(SimulationData), cH2D);
	DailyRuntimeData *dd_g;
	DailyRuntimeData dd = DailyRuntimeData();

	hipMalloc((void **)&dd_g, sizeof(DailyRuntimeData));
	hipMemcpy(dd_g, &dd, sizeof(DailyRuntimeData), cH2D);

	runAlgorithms<<<sd->blocks, sd->threads>>>(gsd, dd_g);

	hipMemcpy(&dd, dd_g, sizeof(DailyRuntimeData), cD2H);
	hipMemcpy(sd, gsd, sizeof(SimulationData), cD2H);
}

__global__ void runAlgorithms(SimulationData *sd, DailyRuntimeData *drd)
{
	__shared__ double commV[1024]; // block size

	update_statuses(sd, commV);
	reduce(commV);

	infect(sd->population, sd->rand);
	drawStage(sd->population, sd->rgb, sd->populationSize);
}

__device__ void update_statuses(SimulationData *sd, double *cv)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);
	int bid = blockIdx.y * blockDim.x + blockIdx.x;

	Individual individual = sd->population[tid];
	Community i_community = sd->communities[bid];
	hiprandState lcu = sd->rand[tid];

	Virus virus = *sd->virus;

	double individual_v = 0;

	// social activity modifier for indiv based on age
	// after 14 and till 42 this modifier > 0.9
	double daily_a = -0.0005 * pow(individual.age - 28, 2) + 1;

	switch (individual.status)
	{
	case -1: // dead
		break;
	case 0: // healthy
		break;
	case 1: // infected
		individual_v = 1 + daily_a * (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) * virus.ntr;
		individual.state -= 1;
		if (individual.state >= 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus.illness_period);
		}
		break;
	case 2: // ill
		// x0.5 because the person is ill and expiriences symptoms reduced number of contacts
		individual_v = 0.5 * daily_a * (i_community.sdf * tnormal(&lcu, {3, 5, 0, 20})) * virus.ntr;
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus.recovery_period);
		}
		break;
	case 3: // recovering
		individual_v = 0.2 * daily_a * (i_community.sdf * tnormal(&lcu, {5, 5, 0, 20})) * virus.ntr;
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, {90, 10, 60, 130});
		}
		break;
	case 4: // immune
	case 5:
		individual.state -= 1;
		if (individual.state == 0)
			individual.status = 0;
		break;
	default: // super-human
		break;
	}

	cv[threadIdx.x + threadIdx.y * blockDim.x] = individual_v;
	__syncthreads();
	sd->rand[tid] = lcu;
	sd->population[tid] = individual;
	sd->communities[bid] = i_community;
}

__device__ void infect(Individual *pop, hiprandState *rand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual in = pop[tid];
	hiprandState loc = rand[tid];

	// people at age of 18 are least vulnerable. and the age multiplier grows non-lineraly
	// 7.7 for 6yo, same for 30yo. At the age of 50 it is 52. 192.7 for 80yo
	double age_m = (0.05 * pow(in.age - 18, 2) + 0.5);

	double chance = hiprand_normal_double(&loc);
	if (chance > 0.9 && in.status != 4)
	{
		in.status = 1;
		in.state = 3;
	}
	pop[tid] = in;
	rand[tid] = loc;
}

__device__ void drawStage(Individual *population, float3 *rgb, ulong sizePopulation)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	float3 lrgb = rgb[tid];
	if (tid < sizePopulation)
	{
		if (individual.status == 0)
		{ // if not infected, draw as white
			lrgb.x = 1.0;
			lrgb.y = 1.0;
			lrgb.z = 1.0;
		}
		else if (individual.status == -1)
		{ // if dead, draw in black
			lrgb.x = 0.0;
			lrgb.y = 0.0;
			lrgb.z = 0.0;
		}
		else if (individual.status == 1)
		{ // if in infected stage, draw as red
			lrgb.x = 1.0;
			lrgb.y = 0.0;
			lrgb.z = 0.0;
		}
		else if (individual.status == 2)
		{ // if in ill, draw as green
			lrgb.x = 0.0;
			lrgb.y = 1.0;
			lrgb.z = 0.0;
		}
		else if (individual.status == 3)
		{ // if in recovering, draw as blue
			lrgb.x = 0.0;
			lrgb.y = 0.0;
			lrgb.z = 1.0;
		}
		else if (individual.status == 4)
		{ // if in immune, draw as violet
			lrgb.x = 0.7;
			lrgb.y = 0.0;
			lrgb.z = 1.0;
		}
		else if (individual.status == 5)
		{ // if in vaccinated, draw as pink
			lrgb.x = 1.0;
			lrgb.y = 0.7;
			lrgb.z = 0.8;
		}
		rgb[tid] = lrgb;
	}
}