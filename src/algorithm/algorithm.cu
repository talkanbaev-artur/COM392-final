#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
#include "virus.h"
#include "../random.cuh"

void runDay(SimulationData sd, int day) {}

__global__ void runAlgorithms(SimulationData sd) {}

__device__ void update_statuses(Individual *population, Virus *virus, hiprandState *rand) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	
	switch (individual.status) {
		case -1: // dead
			break;
		case 0: // healthy
			break;
		case 1: // infected
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = tnormal(rand, virus->illness_period);
			break;
		case 2: // ill
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = tnormal(rand, virus->recovery_period);
			break;
		case 3: // recovering
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = 100;
			break;
		case 4: // immune
			individual.state -= 1;
			if (individual.state == 0) individual.status = 0;
			break;
		default: // super-human
			break;
	}

}
__device__ void infect() {}
