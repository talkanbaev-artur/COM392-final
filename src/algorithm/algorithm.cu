#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
#include "virus.h"
#include "community.h"
#include "../random.cuh"

void runDay(SimulationData sd, int day)
{
	DailyRuntimeData *dd_g;
	DailyRuntimeData dd = DailyRuntimeData();

	hipMalloc((void **)&dd_g, sizeof(DailyRuntimeData));
<<<<<<< HEAD

	runAlgorithms<<<sd.blocks, sd.threads>>>(sd, dd_g);

	hipMemcpy(&dd, dd_g, sizeof(DailyRuntimeData), cD2H);
}

__global__ void runAlgorithms(SimulationData sd, DailyRuntimeData *drd)
{
	update_statuses(sd.population, sd.virus, sd.community, sd.rand);
=======

	runAlgorithms<<<sd.blocks, sd.threads>>>(sd, dd_g);

	hipMemcpy(&dd, dd_g, sizeof(DailyRuntimeData), cD2H);
}

__global__ void runAlgorithms(SimulationData sd, DailyRuntimeData *drd)
{
	update_statuses(sd.population, sd.virus, sd.rand);
>>>>>>> e6a0e2a084b0f49145a5903436d8810ab1f8c8c1
}

__device__ void update_statuses(Individual *population, Virus *virus, Community *community, hiprandState *rand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	Community i_community = community[blockIdx.y * blockDim.y + blockIdx.x];
	hiprandState lcu = rand[tid];
	float individual_v;

	switch (individual.status)
	{
	case -1: // dead
		break;
	case 0: // healthy
		break;
	case 1: // infected
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->illness_period);
		}
		break;
	case 2: // ill
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->recovery_period);
		}
		break;
	case 3: // recovering
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = 100;
		}
		break;
	case 4: // immune
	case 5:
		individual.state -= 1;
		if (individual.state == 0)
			individual.status = 0;
		break;
	default: // super-human
		break;
	}

	rand[tid] = lcu;
	population[tid] = individual;
	community[blockIdx.y * blockDim.y + blockIdx.x] = i_community;
}

__device__ void infect() {}
