#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
#include "virus.h"
#include "community.h"
#include "../random.cuh"

void runDay(SimulationData *sd, int day)
{
	SimulationData *gsd;
	hipMalloc((void **)&gsd, sizeof(SimulationData));
	hipMemcpy(gsd, sd, sizeof(SimulationData), cH2D);
	DailyRuntimeData *dd_g;
	DailyRuntimeData dd = DailyRuntimeData();

	hipMalloc((void **)&dd_g, sizeof(DailyRuntimeData));

	runAlgorithms<<<sd->blocks, sd->threads>>>(gsd, dd_g);

	hipMemcpy(&dd, dd_g, sizeof(DailyRuntimeData), cD2H);
	hipMemcpy(sd, gsd, sizeof(SimulationData), cD2H);
}

__global__ void runAlgorithms(SimulationData *sd, DailyRuntimeData *drd)
{
	update_statuses(sd->population, sd->virus, sd->communities, sd->rand);
	infect(sd->population, sd->rand);
	drawStage(sd->population, sd->rgb, sd->populationSize);
}

__device__ void update_statuses(Individual *population, Virus *virus, Community *communities, hiprandState *rand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	Community i_community = communities[blockIdx.y * blockDim.y + blockIdx.x];
	hiprandState lcu = rand[tid];

	float individual_v;
	// individual.susceptibility + (individual.age/100) + (community.sdf * individual.daily_contacts) + virus.nrt

	switch (individual.status)
	{
	case -1: // dead
		individual_v = 0;
		break;
	case 0: // healthy
		individual_v = individual.susceptibility + (individual.age / 100) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		break;
	case 1: // infected
		individual_v = individual.susceptibility + (individual.age / 20) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state >= 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->illness_period);
		}
		break;
	case 2: // ill
		individual_v = individual.susceptibility + (individual.age / 10) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->recovery_period);
		}
		break;
	case 3: // recovering
		individual_v = individual.susceptibility + (individual.age / 50) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = 100;
		}
		break;
	case 4: // immune
	case 5:
		individual_v = (individual.susceptibility / 10) + (individual.age / 100) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state == 0)
			individual.status = 0;
		break;
	default: // super-human
		break;
	}

	rand[tid] = lcu;
	population[tid] = individual;
	communities[blockIdx.y * blockDim.y + blockIdx.x] = i_community;
}

__device__ void infect(Individual *pop, hiprandState *rand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual in = pop[tid];
	hiprandState loc = rand[tid];
	double chance = hiprand_normal_double(&loc);
	if (chance > 0.9 && in.status != 4)
	{
		in.status = 1;
		in.state = 3;
	}
	pop[tid] = in;
	rand[tid] = loc;
}

__device__ void drawStage(Individual *population, float3 *rgb, ulong sizePopulation)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	float3 lrgb = rgb[tid];
	if (tid < sizePopulation)
	{
		if (individual.status == 0)
		{ // if not infected, draw as white
			lrgb.x = 1.0;
			lrgb.y = 1.0;
			lrgb.z = 1.0;
		}
		else if (individual.status == -1)
		{ // if dead, draw in black
			lrgb.x = 0.0;
			lrgb.y = 0.0;
			lrgb.z = 0.0;
		}
		else if (individual.status == 1)
		{ // if in infected stage, draw as red
			lrgb.x = 1.0;
			lrgb.y = 0.0;
			lrgb.z = 0.0;
		}
		else if (individual.status == 2)
		{ // if in ill, draw as green
			lrgb.x = 0.0;
			lrgb.y = 1.0;
			lrgb.z = 0.0;
		}
		else if (individual.status == 3)
		{ // if in recovering, draw as blue
			lrgb.x = 0.0;
			lrgb.y = 0.0;
			lrgb.z = 1.0;
		}
		else if (individual.status == 4)
		{ // if in immune, draw as violet
			lrgb.x = 0.7;
			lrgb.y = 0.0;
			lrgb.z = 1.0;
		}
		else if (individual.status == 5)
		{ // if in vaccinated, draw as pink
			lrgb.x = 1.0;
			lrgb.y = 0.7;
			lrgb.z = 0.8;
		}
		rgb[tid] = lrgb;
	}
}