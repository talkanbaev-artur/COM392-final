#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
#include "virus.h"
#include "community.h"
#include "../random.cuh"

void runDay(SimulationData sd, int day)
{
	DailyRuntimeData *dd_g;
	DailyRuntimeData dd = DailyRuntimeData();

	hipMalloc((void **)&dd_g, sizeof(DailyRuntimeData));
<<<<<<< HEAD

	runAlgorithms<<<sd.blocks, sd.threads>>>(sd, dd_g);

	hipMemcpy(&dd, dd_g, sizeof(DailyRuntimeData), cD2H);
}

__global__ void runAlgorithms(SimulationData sd, DailyRuntimeData *drd)
{
	update_statuses(sd.population, sd.virus, sd.community, sd.rand);
=======

	runAlgorithms<<<sd.blocks, sd.threads>>>(sd, dd_g);

	hipMemcpy(&dd, dd_g, sizeof(DailyRuntimeData), cD2H);
}

__global__ void runAlgorithms(SimulationData sd, DailyRuntimeData *drd)
{
	update_statuses(sd.population, sd.virus, sd.rand);
>>>>>>> e6a0e2a084b0f49145a5903436d8810ab1f8c8c1
}

__device__ void update_statuses(Individual *population, Virus *virus, Community *community, hiprandState *rand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	Community i_community = community[blockIdx.y * blockDim.y + blockIdx.x];
	hiprandState lcu = rand[tid];
	float individual_v;

	switch (individual.status)
	{
	case -1: // dead
		break;
	case 0: // healthy
		break;
	case 1: // infected
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->illness_period);
		}
		break;
	case 2: // ill
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->recovery_period);
		}
		break;
	case 3: // recovering
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = 100;
		}
		break;
	case 4: // immune
	case 5:
		individual.state -= 1;
		if (individual.state == 0)
			individual.status = 0;
		break;
	default: // super-human
		break;
	}

	rand[tid] = lcu;
	population[tid] = individual;
	community[blockIdx.y * blockDim.y + blockIdx.x] = i_community;
}

__device__ void infect() {}

__device__ void drawStage(Individual *population, float3 *rgb, ulong sizePopulation) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	float3 lrgb = rgb;
	if (tid < sizePopulation){
		if (individual.status == 0) {  // if not infected, draw as white
			lrgb[tid].x = 1.0;
			lrgb[tid].y = 1.0;
			lrgb[tid].z = 1.0;
		}
		else if (individual.status == -1) {  // if dead, draw in black
			lrgb[tid].x = 0.0;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 0.0;
		}
		else if (individual.status < 1) { // if in infected stage, draw as red
			lrgb[tid].x = 1.0;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 0.0;
		}
		else if (individual.status < 2) { // if in ill, draw as green
			lrgb[tid].x = 0.0;
			lrgb[tid].y = 1.0;
			lrgb[tid].z = 0.0;
		}
		else if (individual.status < 3) { // if in recovering, draw as blue
			lrgb[tid].x = 0.0;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 1.0;
		}
		else if (individual.status < 4) { // if in immune, draw as violet
			lrgb[tid].x = 0.7;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 1.0;
		}
		else if (individual.status < 5) { // if in vaccinated, draw as pink
			lrgb[tid].x = 1.0;
			lrgb[tid].y = 0.7;
			lrgb[tid].z = 0.8;
		}
	}
}