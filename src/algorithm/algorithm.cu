#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
<<<<<<< HEAD
#include "virus.h"
#include "../random.cuh"
=======

__device__ void progressInfection(Individual individual);

__device__ void progressImmunity(Individual individual);
>>>>>>> 1b1d8df36116cacb26bc415f09d92a2bc4355ce8

void runDay(SimulationData sd, int day) {}

__global__ void runAlgorithms(SimulationData sd) {}

<<<<<<< HEAD
__device__ void update_statuses(Individual *population, Virus *virus, hiprandState *rand) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	
	switch (individual.status) {
		case -1: // dead
			break;
		case 0: // healthy
			break;
		case 1: // infected
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = tnormal(rand, virus->illness_period);
			break;
		case 2: // ill
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = tnormal(rand, virus->recovery_period);
			break;
		case 3: // recovering
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = 100;
			break;
		case 4: // immune
			individual.state -= 1;
			if (individual.state == 0) individual.status = 0;
			break;
		default: // super-human
			break;
	}

}
=======
__device__ void update_statuses(Individual *population) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	
	switch (individual.status) {
		case -1: // dead
			break;
		case 0: // healthy
			break;
		case 1: // infected
			progressInfection(individual);
			break;
		case 2: // immune
			progressImmunity(individual);
			break;
		default: // super-human
			break;
	}

}

__device__ void progressInfection(Individual individual){
	individual.state -= 1;
	if (individual.state == 0) individual.status = 0;
}

__device__ void progressImmunity(Individual individual){
	individual.state -= 1;
	if (individual.state == 0) individual.status = 0;
}

>>>>>>> 1b1d8df36116cacb26bc415f09d92a2bc4355ce8
__device__ void infect() {}
