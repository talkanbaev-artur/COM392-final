#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
<<<<<<< HEAD
<<<<<<< HEAD
#include "virus.h"
#include "../random.cuh"
=======
=======
>>>>>>> 0b8d9774c8ba05d1e5bea48d7f2b8e11735d6b47

__device__ void progressInfection(Individual individual);

__device__ void progressImmunity(Individual individual);
<<<<<<< HEAD
>>>>>>> 1b1d8df36116cacb26bc415f09d92a2bc4355ce8
=======
>>>>>>> 0b8d9774c8ba05d1e5bea48d7f2b8e11735d6b47

void runDay(SimulationData sd, int day) {}

__global__ void runAlgorithms(SimulationData sd) {}

<<<<<<< HEAD
<<<<<<< HEAD
__device__ void update_statuses(Individual *population, Virus *virus, hiprandState *rand) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	
	switch (individual.status) {
		case -1: // dead
			break;
		case 0: // healthy
			break;
		case 1: // infected
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = tnormal(rand, virus->illness_period);
			break;
		case 2: // ill
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = tnormal(rand, virus->recovery_period);
			break;
		case 3: // recovering
			individual.state -= 1;
			if (individual.state == 0) individual.status++;
			individual.state = 100;
			break;
		case 4: // immune
			individual.state -= 1;
			if (individual.state == 0) individual.status = 0;
			break;
		default: // super-human
			break;
	}

}
=======
=======
>>>>>>> 0b8d9774c8ba05d1e5bea48d7f2b8e11735d6b47
__device__ void update_statuses(Individual *population) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	
	switch (individual.status) {
		case -1: // dead
			break;
		case 0: // healthy
			break;
		case 1: // infected
			progressInfection(individual);
			break;
		case 2: // immune
			progressImmunity(individual);
			break;
		default: // super-human
			break;
	}

}

__device__ void progressInfection(Individual individual){
	individual.state -= 1;
	if (individual.state == 0) individual.status = 0;
}

__device__ void progressImmunity(Individual individual){
	individual.state -= 1;
	if (individual.state == 0) individual.status = 0;
}

<<<<<<< HEAD
>>>>>>> 1b1d8df36116cacb26bc415f09d92a2bc4355ce8
=======
>>>>>>> 0b8d9774c8ba05d1e5bea48d7f2b8e11735d6b47
__device__ void infect() {}
