#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"
#include "virus.h"
#include "community.h"
#include "../random.cuh"

void runDay(SimulationData sd, int day)
{
	DailyRuntimeData *dd_g;
	DailyRuntimeData dd = DailyRuntimeData();

	hipMalloc((void **)&dd_g, sizeof(DailyRuntimeData));

	runAlgorithms<<<sd.blocks, sd.threads>>>(sd, dd_g);

	hipMemcpy(&dd, dd_g, sizeof(DailyRuntimeData), cD2H);
}

__global__ void runAlgorithms(SimulationData sd, DailyRuntimeData *drd)
{
	update_statuses(sd.population, sd.virus, sd.communities, sd.rand);
}

__device__ void update_statuses(Individual *population, Virus *virus, Community *communities, hiprandState *rand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	Community i_community = communities[blockIdx.y * blockDim.y + blockIdx.x];
	hiprandState lcu = rand[tid];

	float individual_v;
	// individual.susceptibility + (individual.age/100) + (community.sdf * individual.daily_contacts) + virus.nrt

	switch (individual.status)
	{
	case -1: // dead
		individual_v = 0;
		break;
	case 0: // healthy
		individual_v = individual.susceptibility + (individual.age/100) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		break;
	case 1: // infected
		individual_v = individual.susceptibility + (individual.age/20) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->illness_period);
		}
		break;
	case 2: // ill
		individual_v = individual.susceptibility + (individual.age/10) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = tnormal(&lcu, virus->recovery_period);
		}
		break;
	case 3: // recovering
		individual_v = individual.susceptibility + (individual.age/50) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state == 0)
		{
			individual.status++;
			individual.state = 100;
		}
		break;
	case 4: // immune
	case 5:
		individual_v = (individual.susceptibility/10) + (individual.age/100) + (i_community.sdf * tnormal(&lcu, individual.daily_contacts)) + virus->ntr;
		individual.state -= 1;
		if (individual.state == 0)
			individual.status = 0;
		break;
	default: // super-human
		break;
	}

	rand[tid] = lcu;
	population[tid] = individual;
	communities[blockIdx.y * blockDim.y + blockIdx.x] = i_community;
}

__device__ void infect() {}

__device__ void drawStage(Individual *population, float3 *rgb, ulong sizePopulation) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	float3 lrgb = rgb;
	if (tid < sizePopulation){
		if (individual.status == 0) {  // if not infected, draw as white
			lrgb[tid].x = 1.0;
			lrgb[tid].y = 1.0;
			lrgb[tid].z = 1.0;
		}
		else if (individual.status == -1) {  // if dead, draw in black
			lrgb[tid].x = 0.0;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 0.0;
		}
		else if (individual.status < 1) { // if in infected stage, draw as red
			lrgb[tid].x = 1.0;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 0.0;
		}
		else if (individual.status < 2) { // if in ill, draw as green
			lrgb[tid].x = 0.0;
			lrgb[tid].y = 1.0;
			lrgb[tid].z = 0.0;
		}
		else if (individual.status < 3) { // if in recovering, draw as blue
			lrgb[tid].x = 0.0;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 1.0;
		}
		else if (individual.status < 4) { // if in immune, draw as violet
			lrgb[tid].x = 0.7;
			lrgb[tid].y = 0.0;
			lrgb[tid].z = 1.0;
		}
		else if (individual.status < 5) { // if in vaccinated, draw as pink
			lrgb[tid].x = 1.0;
			lrgb[tid].y = 0.7;
			lrgb[tid].z = 0.8;
		}
	}
}