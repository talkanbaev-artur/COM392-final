#include "hip/hip_runtime.h"
#include "algorithm.h"
#include "individual.h"

__device__ void progressInfection(Individual individual);

__device__ void progressImmunity(Individual individual);

void runDay(SimulationData sd, int day) {}

__global__ void runAlgorithms(SimulationData sd) {}

__device__ void update_statuses(Individual *population) {
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	Individual individual = population[tid];
	
	switch (individual.status) {
		case -1: // dead
			break;
		case 0: // healthy
			break;
		case 1: // infected
			progressInfection(individual);
			break;
		case 2: // immune
			progressImmunity(individual);
			break;
		default: // super-human
			break;
	}

}

__device__ void progressInfection(Individual individual){
	individual.state -= 1;
	if (individual.state == 0) individual.status = 0;
}

__device__ void progressImmunity(Individual individual){
	individual.state -= 1;
	if (individual.state == 0) individual.status = 0;
}

__device__ void infect() {}
