#include "hip/hip_runtime.h"
#include "simulationData.h"
#include <stdio.h>
DailyRuntimeData::DailyRuntimeData(/* args */)
{
	s = i = r = 0;
	gV = 0;
}

DailyRuntimeData::~DailyRuntimeData()
{
}

SimulationData::SimulationData(Params p)
{
	printf("Starting simulation data intialisation process...\n");
	this->populationSize = p.getPopSize();

	//we use the default 32x32 thread block size which gives max 1024 tpb.
	//it would be usefull to use 30x32 to fit this pattern into full hd samples
	this->threads.x = 32;
	this->threads.y = 32;
	//round up the number of blocks to fit all data
	this->blocks.x = (p.getWidth() + 31) / 32;
	this->blocks.y = (p.getHeight() + 31) / 32;

	long commMemSize = this->blocks.x * this->blocks.y * sizeof(Community);

	hipError_t err;

	err = hipMalloc((void **)&(this->rand), this->populationSize * sizeof(hiprandState));
	if (err != hipSuccess)
	{
		printf("cuda error allocating random = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->communities), commMemSize);
	if (err != hipSuccess)
	{
		printf("cuda error allocating communities memory = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->virus), sizeof(Virus));
	if (err != hipSuccess)
	{
		printf("cuda error allocating virus memory = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->population), this->populationSize * sizeof(Individual));
	if (err != hipSuccess)
	{
		printf("cuda error allocating population memory = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->rgb), this->populationSize * sizeof(float3));
	if (err != hipSuccess)
	{
		printf("cuda error allocating texture rgb map = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	int *res, res2;
	hipMalloc((void **)&res, sizeof(int));

	initialiseCuRand<<<blocks, threads>>>(this->populationSize, this->rand);
	initialisePopulation<<<blocks, threads>>>(populationSize, p, population, rand);
	initVirus<<<1, 1>>>(p, virus);
	initCommunities<<<blocks, 1>>>(blocks.x * blocks.y, communities);

	hipMemcpy(&res2, res, sizeof(int), cD2H);
	printf("Simulation data successfully initialised\n");
}

__global__ void initialiseCuRand(int population, hiprandState *hiprand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	if (tid < population)
	{
		hiprand_init(1, tid, 0, &hiprand[tid]);
	}
}

__global__ void initialisePopulation(int population, Params p, Individual *people, hiprandState *c)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	if (tid < population)
	{
		Individual i(p, &c[tid]);
		people[tid] = i;
	}
}

__global__ void initVirus(Params p, Virus *v)
{
	Virus vl(p);
	*v = vl;
}
__global__ void initCommunities(int comNum, Community *c)
{
	int bid = blockIdx.y * blockDim.y + blockIdx.x;
	if (bid < comNum)
	{
		Community c_l;
		c[bid] = c_l;
	}
}

SimulationData::~SimulationData()
{
	hipFree(this->rgb);
	hipFree(this->rand);
	hipFree(this->population);
	hipFree(this->communities);
	hipFree(this->virus);
	printf("Successfully finished simulation data lifecycle\n");
}