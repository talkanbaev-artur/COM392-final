#include "hip/hip_runtime.h"
#include "simulationData.h"
#include <stdio.h>
DailyRuntimeData::DailyRuntimeData(/* args */)
{
}

DailyRuntimeData::~DailyRuntimeData()
{
}

SimulationData::SimulationData(Params p)
{
	printf("Starting simulation data intialisation process...\n");
	this->populationSize = p.getPopSize();

	//we use the default 32x32 thread block size which gives max 1024 tpb.
	//it would be usefull to use 30x32 to fit this pattern into full hd samples
	this->threads.x = 32;
	this->threads.y = 32;
	//round up the number of blocks to fit all data
	this->blocks.x = (p.getWidth() + 31) / 32;
	this->blocks.y = (p.getHeight() + 31) / 32;

	long commMemSize = this->blocks.x * this->blocks.y * sizeof(Community);

	hipError_t err;

	err = hipMalloc((void **)&(this->rand), this->populationSize * sizeof(hiprandState));
	if (err != hipSuccess)
	{
		printf("cuda error allocating random = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->communities), commMemSize);
	if (err != hipSuccess)
	{
		printf("cuda error allocating communities memory = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->virus), sizeof(Virus));
	if (err != hipSuccess)
	{
		printf("cuda error allocating virus memory = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->population), this->populationSize * sizeof(Individual));
	if (err != hipSuccess)
	{
		printf("cuda error allocating population memory = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc((void **)&(this->rgb), this->populationSize * sizeof(float3));
	if (err != hipSuccess)
	{
		printf("cuda error allocating texture rgb map = %s\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	initialiseCuRand<<<blocks, threads>>>(this->populationSize, this->rand);

	printf("Simulation data successfully initialised\n");
}

__global__ void initialiseCuRand(int population, hiprandState *hiprand)
{
	int x = threadIdx.x + (blockIdx.x * blockDim.x);
	int y = threadIdx.y + (blockIdx.y * blockDim.y);
	int tid = x + (y * blockDim.x * gridDim.x);

	if (tid < population)
		hiprand_init(tid, 0, 0, &hiprand[tid]);
}

SimulationData::~SimulationData()
{
	hipFree(this->rgb);
	hipFree(this->rand);
	hipFree(this->population);
	hipFree(this->communities);
	hipFree(this->virus);
}