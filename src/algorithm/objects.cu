#include "individual.h"
#include "virus.h"
#include "community.h"
#include "../random.cuh"

__device__ Individual::Individual(Params *p, hiprandState *rand)
{
	this->status = 0;
	this->state = 0;
	this->immunity = 0;
	this->immunity_q = 0;
	this->vaccination_h = 0;
	const struct nd_value def_suc = {0.5, 1};
	const struct tnd_value def_age = {40.0, 5, 10.0, 80.0};
	this->susceptibility = normal(rand, def_suc);
	this->age = tnormal(rand, def_age);
	this->daily_contacts = def_age;
}

__device__ Individual::~Individual()
{
}

__device__ Virus::Virus(Params p)
{
	env_factor = p.virusEnvSupport;
	ntr = p.virusNtr;
	incubation_period = {7.0, 1.3, 2.0, 14.0};
	illness_period = {3.0, 1.0, 1.0, 5.0};
	recovery_period = {2.0, 0.2, 1.0, 3.0};
	cfr = {0.5, 0.33};
}

__device__ Virus::~Virus()
{
}

__device__ Community::Community()
{
	igi = 0.0;
	sdf = 1.0;
}
__device__ Community::~Community() {}
