#include "individual.h"
#include "virus.h"
#include "community.h"
#include "../random.cuh"

__device__ Individual::Individual(Params p, hiprandState *rand)
{
	this->status = 0;
	this->state = 0;
	this->immunity = 0;
	this->immunity_q = 0;
	this->vaccination_h = 0;
	const struct nd_value def_suc = {0.6, 0.12};
	const struct tnd_value def_age = {29, 13, 6, 80};
	hiprandState n_rand = *rand;
	this->susceptibility = normal(&n_rand, def_suc);
	this->age = tnormal(&n_rand, def_age);
	this->daily_contacts = {40, 150, 4, 2000};
	rand = &n_rand;
}

__device__ Individual::~Individual()
{
}

__device__ Virus::Virus(Params p)
{
	env_factor = p.virusEnvSupport;
	ntr = p.virusNtr;
	incubation_period = {14.0, 1.3, 2.0, 20.0};
	illness_period = {3.0, 1.0, 1.0, 5.0};
	recovery_period = {2.0, 0.6, 1.0, 3.0};
	cfr = {0.5, 0.33};
}

__device__ Virus::~Virus()
{
}

__device__ Community::Community()
{
	igi = 0.0;
	sdf = 1.0;
}
__device__ Community::~Community() {}
