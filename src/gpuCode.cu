#include "hip/hip_runtime.h"
/*******************************************************************************
*
*   COMMENTS GO HERE
*
*   TODO LIST GOES HERE
*
*******************************************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpuCode.h"
#include "params.h"

texture<float, 2> texBlue;

/******************************************************************************/
// VIRUS SIMULATION CODE
/******************************************************************************/
GPU_Palette initPopulation(void) // for simulating virus
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = 32;  // 32 x 32 = 1024 blocks
  X.gBlocks.y = 32;
  X.gBlocks.z = 1;

  X.palette_width = 1024;       // save this info
  X.palette_height = 1024;
  X.num_pixels = 1024*1024; // 1048576
  X.memSize =  1024*1024 * sizeof(float);
  X.memIntSize =  1024*1024 * sizeof(int);

  // keep color stuff for visualizing virus spread
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.green, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.blue, X.memSize);  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.rand, X.num_pixels * sizeof(hiprandState));
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

  // for initializing population with (random) susceptibility ratings
  err = hipMalloc((void**) &X.susc, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating susc = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.stage, X.memIntSize);
  if(err != hipSuccess){
    printf("cuda error allocating stage = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.ming, X.memIntSize);
  if(err != hipSuccess){
    printf("cuda error allocating ming = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }


  initRands <<< X.gBlocks, X.gThreads >>> (X.rand, time(NULL), X.num_pixels);

  hipChannelFormatDesc desc= hipCreateChannelDesc <float>();
  unsigned int pitch = sizeof(float)*1024;
  hipBindTexture2D(NULL, texBlue, X.blue, desc, 1024, 1024, pitch);

  // set reds, greens, and blues to zero
  setMap <<< X.gBlocks, X.gThreads >>> (X.red, 0.0, X.num_pixels);
  setMap <<< X.gBlocks, X.gThreads >>> (X.green, 0.0, X.num_pixels);
  setMap <<< X.gBlocks, X.gThreads >>> (X.blue, 0.0, X.num_pixels);

  return X;
}

/******************************************************************************/
// analogous to updatePalette in runmode 1
int updatePopulation(GPU_Palette* P, AParams* PARAMS, int day){

  // 1) have people mingle, some will come in contact with contageous people
  //    where infection status will go from 'not infected' to
  //    'contageous' based on dice roll and spread rate
  // 2) after period of time, have people go from contageous to recovery
  // 3) after another period, go from recovery to either immune or to death
  //    based on susceptibility and deadliness of virus
  // 4) if want to visualize, write a kernel that updates colors based on stage
  // 5) return how many people die in the simulation, better yet, track
  //    infection waves over time

  // place-holder code, just draw the population every day over ten years;
  // whole screen goes from black to white over time.
  float goo = day/3650.0;
  setMap <<< P->gBlocks, P->gThreads >>> (P->red, goo, P->num_pixels);
  setMap <<< P->gBlocks, P->gThreads >>> (P->green, goo, P->num_pixels);
  setMap <<< P->gBlocks, P->gThreads >>> (P->blue, goo, P->num_pixels);

  return 0;
}

/******************************************************************************/
__global__ void setMap(float* map, float val, long sizePopulation){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  if (tid < sizePopulation){
    map[tid] = val;
  }
}

/******************************************************************************/
//__global__ void updateReds(float* red){
__global__ void updateReds(float* red, hiprandState* gRand){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  // generate noise
  hiprandState localState = gRand[tid];
  float theRand = hiprand_uniform(&localState); // value between 0-1
//  float theRand = hiprand_poisson(&localState, .5);
  gRand[tid] = localState;

  // sparkle the reds:
  if(theRand > .999) red[tid] = red[tid] *.9;
  else if(theRand < .001) red[tid] = (1.0-red[tid]);
}

/******************************************************************************/
__global__ void updateGreens(float* green){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  green[tid] = green[tid] *.888;
//  green[tid] = green[tid] * 0;
}

/******************************************************************************/
__global__ void initRands(hiprandState* state, unsigned long seed, unsigned long numPixels){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  if(tid < numPixels) hiprand_init(seed, tid, 0, &state[tid]);

}

/******************************************************************************/
__global__ void updateBlues(float* blue){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  // find neighborhood average blue value
  float acc = 0.0;
  for (int i = -20; i <= 20; i++){      // 11 pixels-threads in x direction
    for (int j = -20; j <= 20; j++){    // 11 pixels-threads in the y direction
      acc += tex2D(texBlue, x+i, y+j);
    }
  }
  acc /= 241.0;

  blue[tid] = acc;

}