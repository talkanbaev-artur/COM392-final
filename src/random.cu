#include "hip/hip_runtime.h"
#include "random.cuh"
__device__ int tnormal(hiprandState_t *state, tnd_value inps)
{
	int result = static_cast<int>(hiprand_log_normal_double(state, inps.mean, inps.s_deviance) + 0.5);
	return max((int)inps.a, min(result, (int)inps.b));
}

__device__ double normal(hiprandState_t *state, nd_value inps)
{
	return hiprand_log_normal_double(state, inps.mean, inps.s_deviance);
}