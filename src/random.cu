#include "hip/hip_runtime.h"
#include "random.cuh"
__device__ int tnormal(hiprandState_t *state, tnd_value inps)
{
	double result = static_cast<int>(hiprand_normal_double(state) * inps.s_deviance + inps.mean + 0.49);
	return max(inps.a, min(result, inps.b));
}

__device__ double normal(hiprandState_t *state, nd_value inps)
{
	return hiprand_normal_double(state) * inps.s_deviance + inps.mean;
}