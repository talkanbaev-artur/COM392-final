#include "hip/hip_runtime.h"
/*******************************************************************************
*
*   COMMENTS GO HERE
*
*   TODO LIST GOES HERE
*
*******************************************************************************/
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpuCode.h"
#include "params.h"

texture<float, 2> texBlue;

/******************************************************************************/
// VIRUS SIMULATION CODE
/******************************************************************************/
GPU_Palette initPopulation(void) // for simulating virus
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = 32;  // 32 x 32 = 1024 blocks
  X.gBlocks.y = 32;
  X.gBlocks.z = 1;

  X.palette_width = 1024;       // save this info
  X.palette_height = 1024;
  X.num_pixels = 1024*1024; // 1048576
  X.memSize =  1024*1024 * sizeof(float);
  X.memIntSize =  1024*1024 * sizeof(int);

  // keep color stuff for visualizing virus spread
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.green, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.blue, X.memSize);  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.rand, X.num_pixels * sizeof(hiprandState));
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

  // for initializing population with (random) susceptibility ratings
  err = hipMalloc((void**) &X.susc, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating susc = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.stage, X.memIntSize);
  if(err != hipSuccess){
    printf("cuda error allocating stage = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.ming, X.memIntSize);
  if(err != hipSuccess){
    printf("cuda error allocating ming = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }


  initRands <<< X.gBlocks, X.gThreads >>> (X.rand, time(NULL), X.num_pixels);

  hipChannelFormatDesc desc= hipCreateChannelDesc <float>();
  unsigned int pitch = sizeof(float)*1024;
  hipBindTexture2D(NULL, texBlue, X.blue, desc, 1024, 1024, pitch);

  // set reds, greens, and blues to zero
  setMap <<< X.gBlocks, X.gThreads >>> (X.red, 0.0, X.num_pixels);
  setMap <<< X.gBlocks, X.gThreads >>> (X.green, 0.0, X.num_pixels);
  setMap <<< X.gBlocks, X.gThreads >>> (X.blue, 0.0, X.num_pixels);

  return X;
}

/******************************************************************************/
// analogous to updatePalette in runmode 1
int updatePopulation(GPU_Palette* P, AParams* PARAMS, int day){

  // 1) have people mingle, some will come in contact with contageous people
  //    where infection status will go from 'not infected' to
  //    'contageous' based on dice roll and spread rate
  // 2) after period of time, have people go from contageous to recovery
  // 3) after another period, go from recovery to either immune or to death
  //    based on susceptibility and deadliness of virus
  // 4) if want to visualize, write a kernel that updates colors based on stage
  // 5) return how many people die in the simulation, better yet, track
  //    infection waves over time

  // place-holder code, just draw the population every day over ten years;
  // whole screen goes from black to white over time.
  float goo = day/3650.0;
  setMap <<< P->gBlocks, P->gThreads >>> (P->red, goo, P->num_pixels);
  setMap <<< P->gBlocks, P->gThreads >>> (P->green, goo, P->num_pixels);
  setMap <<< P->gBlocks, P->gThreads >>> (P->blue, goo, P->num_pixels);

  return 0;
}

/******************************************************************************/
__global__ void setMap(float* map, float val, long sizePopulation){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  if (tid < sizePopulation){
    map[tid] = val;
  }
}










/******************************************************************************/
// RUNMODE 0 CODE
/******************************************************************************/
// return information about CUDA GPU devices on this machine
int probeGPU(){

  hipError_t err;
  err = hipDeviceReset();

  hipDeviceProp_t prop;
  int count;
  err = hipGetDeviceCount(&count);
  if(err != hipSuccess){
    printf("problem getting device count = %s\n", hipGetErrorString(err));
    return 1;
    }
  printf("number of GPU devices: %d\n\n", count);

  for (int i = 0; i< count; i++){
    printf("************ GPU Device: %d ************\n\n", i);
    err = hipGetDeviceProperties(&prop, i);
    if(err != hipSuccess){
      printf("problem getting device properties = %s\n", hipGetErrorString(err));
      return 1;
      }

    printf("\tName: %s\n", prop.name);
    printf( "\tCompute capability: %d.%d\n", prop.major, prop.minor);
    printf( "\tClock rate: %d\n", prop.clockRate );
    printf( "\tDevice copy overlap: " );
      if (prop.deviceOverlap)
        printf( "Enabled\n" );
      else
        printf( "Disabled\n" );
    printf( "\tKernel execition timeout: " );
      if (prop.kernelExecTimeoutEnabled)
        printf( "Enabled\n" );
      else
        printf( "Disabled\n" );
    printf( "--- Memory Information for device %d ---\n", i );
    printf("\tTotal global mem: %ld\n", prop.totalGlobalMem );
    printf("\tTotal constant Mem: %ld\n", prop.totalConstMem );
    printf("\tMax mem pitch: %ld\n", prop.memPitch );
    printf( "\tTexture Alignment: %ld\n", prop.textureAlignment );
    printf("\n");
    printf( "\tMultiprocessor count: %d\n", prop.multiProcessorCount );
    printf( "\tShared mem per processor: %ld\n", prop.sharedMemPerBlock );
    printf( "\tRegisters per processor: %d\n", prop.regsPerBlock );
    printf( "\tThreads in warp: %d\n", prop.warpSize );
    printf( "\tMax threads per block: %d\n", prop.maxThreadsPerBlock );
    printf( "\tMax block dimensions: (%d, %d, %d)\n",
                  prop.maxThreadsDim[0],
                  prop.maxThreadsDim[1],
                  prop.maxThreadsDim[2]);
    printf( "\tMax grid dimensions: (%d, %d, %d)\n",
                  prop.maxGridSize[0],
                  prop.maxGridSize[1],
                  prop.maxGridSize[2]);
    printf("\n");
  }

return 0;
}






/******************************************************************************/
// RUNMODE 1 CODE
/******************************************************************************/
int updatePalette(GPU_Palette* P){

  updateReds <<< P->gBlocks, P->gThreads >>> (P->red, P->rand);
  updateGreens <<< P->gBlocks, P->gThreads >>> (P->green);
	updateBlues <<< P->gBlocks, P->gThreads >>> (P->blue);

  return 0;
}

/******************************************************************************/
//__global__ void updateReds(float* red){
__global__ void updateReds(float* red, hiprandState* gRand){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  // generate noise
  hiprandState localState = gRand[tid];
  float theRand = hiprand_uniform(&localState); // value between 0-1
//  float theRand = hiprand_poisson(&localState, .5);
  gRand[tid] = localState;

  // sparkle the reds:
  if(theRand > .999) red[tid] = red[tid] *.9;
  else if(theRand < .001) red[tid] = (1.0-red[tid]);
}

/******************************************************************************/
__global__ void updateGreens(float* green){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  green[tid] = green[tid] *.888;
//  green[tid] = green[tid] * 0;
}

/******************************************************************************/
__global__ void initRands(hiprandState* state, unsigned long seed, unsigned long numPixels){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  if(tid < numPixels) hiprand_init(seed, tid, 0, &state[tid]);

}



/******************************************************************************/
__global__ void updateBlues(float* blue){

  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int tid = x + (y * blockDim.x * gridDim.x);

  // find neighborhood average blue value
  float acc = 0.0;
  for (int i = -20; i <= 20; i++){      // 11 pixels-threads in x direction
    for (int j = -20; j <= 20; j++){    // 11 pixels-threads in the y direction
      acc += tex2D(texBlue, x+i, y+j);
    }
  }
  acc /= 241.0;

  blue[tid] = acc;

}


/******************************************************************************/
GPU_Palette initGPUPalette(unsigned int imageWidth, unsigned int imageHeight)
{
  GPU_Palette X;

  X.gThreads.x = 32;  // 32 x 32 = 1024 threads per block
  X.gThreads.y = 32;
  X.gThreads.z = 1;
  X.gBlocks.x = ceil(imageWidth/32);  // however many blocks needed for image
  X.gBlocks.y = ceil(imageHeight/32);
  X.gBlocks.z = 1;

  X.palette_width = imageWidth;       // save this info
  X.palette_height = imageHeight;
  X.num_pixels = imageWidth * imageHeight;
  X.memSize =  imageWidth * imageHeight * sizeof(float);

  // allocate memory on GPU
  hipError_t err;
  err = hipMalloc((void**) &X.red, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating red = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.green, X.memSize);
  if(err != hipSuccess){
    printf("cuda error allocating green = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }
  err = hipMalloc((void**) &X.blue, X.memSize);  // b
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

  err = hipMalloc((void**) &X.rand, X.num_pixels * sizeof(hiprandState));
  if(err != hipSuccess){
    printf("cuda error allocating blue = %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
    }

  initRands <<< X.gBlocks, X.gThreads >>> (X.rand, time(NULL), X.num_pixels);

  hipChannelFormatDesc desc= hipCreateChannelDesc <float>();
  unsigned int pitch = sizeof(float)*imageWidth;
  hipBindTexture2D(NULL, texBlue, X.blue, desc, imageWidth, imageHeight, pitch);


  return X;
}



/******************************************************************************/
int freeGPUPalette(GPU_Palette* P) {

  // free gpu memory
//  hipFree(P->gray);
  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->blue);

  hipUnbindTexture(texBlue);

  return 0;
}

/*************************************************************************/
